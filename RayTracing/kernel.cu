#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <fstream>

#define MAX_DEPTH 5

inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        exit(-1);
    }
    return result;
}

struct Vec3 {
    float x, y, z;
    __host__ __device__ Vec3 operator+(const Vec3& v) const { return {x + v.x, y + v.y, z + v.z}; }
    __host__ __device__ Vec3 operator-(const Vec3& v) const { return {x - v.x, y - v.y, z - v.z}; }
    __host__ __device__ Vec3 operator*(float scalar) const { return {x * scalar, y * scalar, z * scalar}; }
    __host__ __device__ float dot(const Vec3& v) const { return x * v.x + y * v.y + z * v.z; }
    __host__ __device__ Vec3 normalize() const {
        float len = sqrtf(dot(*this));
        return (len > 0) ? *this * (1.0f / len) : Vec3{0, 0, 0};
    }
};

struct Sphere {
    Vec3 center;
    float radius;
    Vec3 color;
};

struct Light {
    Vec3 position;
    Vec3 intensity;
};

struct Plane {
    Vec3 point;
    Vec3 normal;
    Vec3 color;
};

__device__ bool intersectSphere(const Vec3& rayOrigin, const Vec3& rayDir, const Sphere& sphere, float& t) {
    Vec3 oc = rayOrigin - sphere.center;
    float a = rayDir.dot(rayDir);
    float b = 2.0f * oc.dot(rayDir);
    float c = oc.dot(oc) - sphere.radius * sphere.radius;
    float discriminant = b * b - 4 * a * c;
    if (discriminant < 0) return false;
    t = (-b - sqrtf(discriminant)) / (2.0f * a);
    return t >= 0;
}

__device__ bool intersectPlane(const Vec3& rayOrigin, const Vec3& rayDir, const Plane& plane, float& t) {
    float denom = plane.normal.dot(rayDir);
    if (fabs(denom) > 1e-6) {
        Vec3 p0l0 = plane.point - rayOrigin;
        t = p0l0.dot(plane.normal) / denom;
        return t >= 0;
    }
    return false;
}

__device__ Vec3 calculateLighting(const Vec3& intersection, const Vec3& normal, const Vec3& viewDir, const Vec3& objectColor, Light* lights, int numLights, bool inShadow) {
    constexpr float AMBIENT_COEFFICIENT = 0.1f;
    constexpr float DIFFUSE_COEFFICIENT = 1.0f;
    constexpr float SPECULAR_COEFFICIENT = 0.7f;
    constexpr float SHININESS = 32.0f;

    Vec3 color = objectColor * AMBIENT_COEFFICIENT;

    for (int i = 0; i < numLights; ++i) {
        Vec3 lightDir = (lights[i].position - intersection).normalize();
        if (inShadow) continue;

        // Diffuse
        float diff = fmaxf(normal.dot(lightDir), 0.0f);
        color = color + objectColor * (lights[i].intensity * DIFFUSE_COEFFICIENT * diff);

        // Specular
        Vec3 reflectDir = (normal * (2.0f * normal.dot(lightDir)) - lightDir).normalize();
        float spec = powf(fmaxf(viewDir.dot(reflectDir), 0.0f), SHININESS);
        color = color + lights[i].intensity * SPECULAR_COEFFICIENT * spec;
    }

    return color;
}

__device__ Vec3 TraceRay(const Vec3& rayOrigin, const Vec3& rayDir, Sphere* spheres, int numSpheres, Plane* planes, int numPlanes, Light* lights, int numLights, int depth) {
    if (depth > MAX_DEPTH) return {0.0f, 0.0f, 0.0f};

    float closestT = 1e20f;
    int closestSphere = -1, closestPlane = -1;
    bool hitPlane = false;

    for (int i = 0; i < numSpheres; ++i) {
        float t;
        if (intersectSphere(rayOrigin, rayDir, spheres[i], t) && t < closestT) {
            closestT = t;
            closestSphere = i;
            hitPlane = false;
        }
    }

    for (int i = 0; i < numPlanes; ++i) {
        float t;
        if (intersectPlane(rayOrigin, rayDir, planes[i], t) && t < closestT) {
            closestT = t;
            closestPlane = i;
            hitPlane = true;
        }
    }

    Vec3 intersectionPoint = rayOrigin + rayDir * closestT;
    Vec3 viewDir = -rayDir;
    Vec3 color = {0.0f, 0.0f, 0.0f};

    if (hitPlane && closestPlane != -1) {
        color = calculateLighting(intersectionPoint, planes[closestPlane].normal.normalize(), viewDir, planes[closestPlane].color, lights, numLights, false);
    } else if (closestSphere != -1) {
        Vec3 normal = (intersectionPoint - spheres[closestSphere].center).normalize();
        color = calculateLighting(intersectionPoint, normal, viewDir, spheres[closestSphere].color, lights, numLights, false);
    }

    return color;
}

__global__ void renderKernel(Sphere* spheres, int numSpheres, Plane* planes, int numPlanes, Light* lights, int numLights, unsigned char* image, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float u = float(x) / float(width);
        float v = float(y) / float(height);

        Vec3 rayOrigin = {0.0f, 0.0f, 0.0f};
        Vec3 rayDir = {u - 0.5f, v - 0.5f, -1.0f};
        rayDir = rayDir.normalize();

        Vec3 color = TraceRay(rayOrigin, rayDir, spheres, numSpheres, planes, numPlanes, lights, numLights, 0);

        int pixelIndex = (y * width + x) * 3;
        image[pixelIndex] = (unsigned char)(fminf(color.x, 1.0f) * 255);
        image[pixelIndex + 1] = (unsigned char)(fminf(color.y, 1.0f) * 255);
        image[pixelIndex + 2] = (unsigned char)(fminf(color.z, 1.0f) * 255);
    }
}

void renderScene(Sphere* spheres, int numSpheres, Plane* planes, int numPlanes, Light* lights, int numLights, unsigned char* image, int width, int height) {
    Sphere* d_spheres;
    Plane* d_planes;
    Light* d_lights;
    unsigned char* d_image;

    checkCuda(hipMallocManaged(&d_spheres, sizeof(Sphere) * numSpheres));
    checkCuda(hipMallocManaged(&d_planes, sizeof(Plane) * numPlanes));
    checkCuda(hipMallocManaged(&d_lights, sizeof(Light) * numLights));
    checkCuda(hipMallocManaged(&d_image, sizeof(unsigned char) * width * height * 3));

    hipMemcpy(d_spheres, spheres, sizeof(Sphere) * numSpheres, hipMemcpyHostToDevice);
    hipMemcpy(d_planes, planes, sizeof(Plane) * numPlanes, hipMemcpyHostToDevice);
    hipMemcpy(d_lights, lights, sizeof(Light) * numLights, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    renderKernel<<<gridSize, blockSize>>>(d_spheres, numSpheres, d_planes, numPlanes, d_lights, numLights, d_image, width, height);
    checkCuda(hipDeviceSynchronize());

    hipMemcpy(image, d_image, sizeof(unsigned char) * width * height * 3, hipMemcpyDeviceToHost);

    hipFree(d_spheres);
    hipFree(d_planes);
    hipFree(d_lights);
    hipFree(d_image);
}

void saveBMP(const char* filename, unsigned char* image, int width, int height) {
    std::ofstream file(filename, std::ios::binary);

    unsigned char header[54] = {
        'B', 'M', 0, 0, 0, 0, 0, 0, 0, 0, 54, 0, 0, 0, 40, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 24, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
    };

    int fileSize = 54 + width * height * 3;
    header[2] = (unsigned char)(fileSize);
    header[3] = (unsigned char)(fileSize >> 8);
    header[4] = (unsigned char)(fileSize >> 16);
    header[5] = (unsigned char)(fileSize >> 24);

    header[18] = (unsigned char)(width);
    header[19] = (unsigned char)(width >> 8);
    header[20] = (unsigned char)(width >> 16);
    header[21] = (unsigned char)(width >> 24);

    header[22] = (unsigned char)(height);
    header[23] = (unsigned char)(height >> 8);
    header[24] = (unsigned char)(height >> 16);
    header[25] = (unsigned char)(height >> 24);

    file.write(reinterpret_cast<char*>(header), 54);
    file.write(reinterpret_cast<char*>(image), width * height * 3);
    file.close();
}

int main() {
    const int width = 4000;
    const int height = 4000;

    Sphere spheres[] = {
        {{-2.0f, 0.0f, -6.0f}, 1.0f, {1.0f, 0.0f, 0.0f}},
        {{0.0f, 0.0f, -8.0f}, 1.0f, {0.0f, 1.0f, 0.0f}},
        {{1.5f, 0.0f, -6.0f}, 1.0f, {0.0f, 0.0f, 1.0f}}
    };

    Plane planes[] = {
        {{0.0f, -1.0f, 0.0f}, {0.0f, 1.0f, 0.0f}, {1.0f, 1.0f, 1.0f}},
        {{0.0f, 0.0f, -10.0f}, {0.0f, 0.0f, 1.0f}, {1.0f, 1.0f, 1.0f}}
    };

    Light lights[] = {
        {{-10.0f, 10.0f, 10.0f}, {1.0f, 1.0f, 1.0f}}
    };

    unsigned char* image = new unsigned char[width * height * 3];
    renderScene(spheres, 3, planes, 2, lights, 1, image, width, height);
    saveBMP("output.bmp", image, width, height);
    delete[] image;

    return 0;
}
