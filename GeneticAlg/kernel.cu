#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>
#include <fstream>

__device__ float evaluateFitness(const float* coefficients, const float* pointsX, const float* pointsY, int degree, int numPoints) {
    float mse = 0.0f;
    for (int i = 0; i < numPoints; ++i) {
        float predictedY = 0.0f;
        for (int j = 0; j <= degree; ++j) {
            predictedY += coefficients[j] * powf(pointsX[i], j);
        }
        float error = predictedY - pointsY[i];
        mse += error * error;
    }
    return mse / numPoints;
}

__device__ int tournamentSelection(float* fitness, int populationSize, hiprandState* state) {
    int bestIndividual = -1;
    float bestFitness = FLT_MAX;
    for (int i = 0; i < 10; ++i) {
        int individual = hiprand(state) % populationSize;
        if (fitness[individual] < bestFitness) {
            bestFitness = fitness[individual];
            bestIndividual = individual;
        }
    }
    return bestIndividual;
}

__device__ void crossover(const float* parent1, const float* parent2, float* child, int degree, hiprandState* state) {
    int crossoverPoint = 1 + (hiprand(state) % degree);
    for (int i = 0; i < degree + 1; ++i) {
        if (i < crossoverPoint) child[i] = parent1[i];
        else  child[i] = parent2[i];
    }
}

__device__ void mutate(float* individual, int degree, hiprandState* state) {
    for (int i = 0; i <= degree; ++i) individual[i] += hiprand_uniform(state) - 0.5f;
}

__global__ void geneticAlgorithmKernel(float* population, float* pointsX, float* pointsY, float* fitness, int populationSize, int degree, int numPoints, hiprandState* states) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < populationSize) {
        hiprand_init(1234, idx, 0, &states[idx]);
        fitness[idx] = evaluateFitness(&population[idx * (degree + 1)], pointsX, pointsY, degree, numPoints);
    }
}

__global__ void nextGeneration(float* population, float* newPopulation, float* fitness, int populationSize, int degree, bool applyMutation, hiprandState* states) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < populationSize) {
        hiprandState localState = states[idx];
        int parent1Idx = tournamentSelection(fitness, populationSize, &localState);
        int parent2Idx = tournamentSelection(fitness, populationSize, &localState);
        float* parent1 = &population[parent1Idx * (degree + 1)];
        float* parent2 = &population[parent2Idx * (degree + 1)];

        float* child = &newPopulation[idx * (degree + 1)];
        crossover(parent1, parent2, child, degree, &localState);
        if (applyMutation) {
            mutate(child, degree, &localState);
        }
    }
}

int main() {
    setlocale(LC_ALL, "RU");
    int degree = 4;
    int numPoints = 500;
    int populationSize = 1000;
    int maxGenerations = 1000;

    std::vector<float> hostPointsX(numPoints);
    std::vector<float> hostPointsY(numPoints);
    for (int i = 0; i < numPoints; ++i) {
        hostPointsX[i] = static_cast<float>((i + 1) / 50);
        hostPointsY[i] = 1 * hostPointsX[i] * hostPointsX[i] * hostPointsX[i] * hostPointsX[i] +
            2 * hostPointsX[i] * hostPointsX[i] * hostPointsX[i] +
            3 * hostPointsX[i] * hostPointsX[i] +
            4 * hostPointsX[i] +
            5;
    }
    std::vector<float> hostPopulation(populationSize * (degree + 1));
    for (int i = 0; i < populationSize * (degree + 1); ++i) {
        hostPopulation[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    float* devicePopulation, * devicePointsX, * devicePointsY, * deviceFitness, * deviceNewPopulation;
    hipMalloc(&devicePopulation, populationSize * (degree + 1) * sizeof(float));
    hipMalloc(&deviceNewPopulation, populationSize * (degree + 1) * sizeof(float));
    hipMalloc(&devicePointsX, numPoints * sizeof(float));
    hipMalloc(&devicePointsY, numPoints * sizeof(float));
    hipMalloc(&deviceFitness, populationSize * sizeof(float));
    hipMemcpy(devicePopulation, hostPopulation.data(), populationSize * (degree + 1) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devicePointsX, hostPointsX.data(), numPoints * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devicePointsY, hostPointsY.data(), numPoints * sizeof(float), hipMemcpyHostToDevice);

    hiprandState* deviceStates;
    hipMalloc(&deviceStates, populationSize * sizeof(hiprandState));

    float bestFitness = 1e16f;
    std::vector<float> bestCoefficients(degree + 1);
    float previousBestFitness = 1e16f;
    int generation = 0;
    int repeatCount = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (generation; generation < maxGenerations; ++generation) {
        geneticAlgorithmKernel << <populationSize / 1024+1, 1024 >> > (devicePopulation, devicePointsX, devicePointsY, deviceFitness, populationSize, degree, numPoints, deviceStates);
        hipDeviceSynchronize();
        bool applyMutation = (repeatCount >= 4);

        nextGeneration << <populationSize/1024+1, 1024 >> > (devicePopulation, deviceNewPopulation, deviceFitness, populationSize, degree, applyMutation, deviceStates);
        hipDeviceSynchronize();
        hipMemcpy(hostPopulation.data(), deviceNewPopulation, populationSize * (degree + 1) * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(devicePopulation, deviceNewPopulation, populationSize * (degree + 1) * sizeof(float), hipMemcpyHostToDevice);

        thrust::device_ptr<float> dev_ptr(deviceFitness);
        thrust::device_ptr<float> min_ptr = thrust::min_element(dev_ptr, dev_ptr + populationSize);
        bestFitness = *min_ptr;

        int bestIndex = min_ptr - dev_ptr;
        std::copy(hostPopulation.begin() + bestIndex * (degree + 1), hostPopulation.begin() + (bestIndex + 1) * (degree + 1), bestCoefficients.begin());

        if (bestFitness == previousBestFitness) repeatCount++;
        else repeatCount = 0;
        previousBestFitness = bestFitness;
        if (bestFitness < 0.1) break;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Время выполнения на GPU: " << milliseconds / 1000.0f << " секунд" << std::endl;
    std::cout << "Лучшие коэффициенты полинома: ";
    for (float coeff : bestCoefficients) {
        std::cout << coeff << " ";
    }
    std::cout << std::endl;
    std::cout << "Лучшая приспособленность: " << bestFitness << std::endl;
    std::cout << "Количество итераций: " << generation << std::endl;
    std::ofstream outFile("results.csv");
    outFile << "X,Y,PredictedY\n";
    for (int i = 0; i < numPoints; ++i) {
        float predictedY = 0.0f;
        for (int j = 0; j <= degree; ++j) {
            predictedY += bestCoefficients[j] * powf(hostPointsX[i], j);
        }
        outFile << hostPointsX[i] << "," << hostPointsY[i] << "," << predictedY << "\n";
    }
    outFile.close();
    hipFree(devicePopulation);
    hipFree(deviceNewPopulation);
    hipFree(devicePointsX);
    hipFree(devicePointsY);
    hipFree(deviceFitness);
    hipFree(deviceStates);
    return 0;
}
